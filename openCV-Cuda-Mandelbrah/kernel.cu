#include "hip/hip_runtime.h"
//
//#include "hip/hip_runtime.h"
//#include ""
//#include <hip/hip_runtime.h>
//#include <iostream>
//#include <math.h>
//#include <fstream>
//#include <windows.h> // contains windef.h which has all the bitmap stuff
//#include <stdio.h> // defines FILENAME_MAX
//#include <stdlib.h>
//#include <direct.h>
//#include <opencv2/opencv.hpp>
//#include <opencv2/imgproc/imgproc.hpp>
//#include <opencv2/highgui/highgui.hpp>
//#include <opencv2/core/core.hpp>
//
//using namespace cv;
//using namespace std;
//
//
//// dimensions are hard coded
//#define WIDTH 4096
//#define HEIGHT 4096
//
//
//// this kernel calculates the pixel value for one pixel
//__global__ void mandelbrot(BYTE* imageData, float unitX, float unitY, int max, int pixelWidth)
//{
//	// get the unique thread index
//	// only using 1, 1 grid
//	int row = blockIdx.y * blockDim.y + threadIdx.y;
//	int col = blockIdx.x * blockDim.x + threadIdx.x;
//
//	// offset values so center is 0, 0
//	float offsetWidth = col - (WIDTH / 2);
//	float offsetHeight = row - (HEIGHT / 2);
//
//	// multiply by our units (applies the zoom)
//	float translatedWidth = offsetWidth * unitX;
//	float translatedHeight = offsetHeight * unitY;
//
//	float x = 0, y = 0;
//	int iter = 0;
//
//	int pos = (WIDTH * row) + col; // the position in the pixel data byte array
//
//	// keep iterating until point escapes mandlebrot set
//	while (1)
//	{
//		if (sqrt((x*x) + (y*y)) > 2) // if magnitude is greater than 2
//		{
//			// point has escaped mandlebrot set - paint white
//			imageData[pos * pixelWidth] = (BYTE)255;
//			break;
//		}
//		if (iter == max)
//		{
//			// point is in the mandlebrot set - paint black
//			imageData[pos * pixelWidth] = (BYTE)0;
//			break;
//		}
//
//		// this applies the mandelbrot equation
//		// Zn+1 = Zn^2 + C
//		float temp = ((x*x) - (y*y)) + translatedWidth;
//		y = (2 * x*y) + translatedHeight;
//		x = temp;
//		iter++;
//	}
//}
//
//int main(int argc, char** argv[])
//{
//	printf("Building image data...\n");
//
//	// this is hard coded sadly
//	dim3 grid(256, 256);
//	dim3 block(16, 16);
//
//	int pixelWidth = 1; // in bytes. bmp doesn't really do binary images so 1 byte is minimum
//	int imageSize = WIDTH * HEIGHT * pixelWidth; // in bytes
//
//	// allocate device memory
//	BYTE * imageData_d = NULL;
//	hipMalloc((void **)&imageData_d, imageSize);
//
//	// the interesting stuff in the mandlebrot set occurs between -2,-2 and 2,2
//	float zoomX = 2, zoomY = 2;
//
//	// max iterations
//	// increasing iterations improves image quality but hits performance
//	int max = 1000;
//
//	float unitX = zoomX / (WIDTH / 2);
//	float unitY = zoomY / (HEIGHT / 2);
//
//	// launch kernel on each pixel
//	mandelbrot <<<grid, block >>> (imageData_d, unitX, unitY, max, pixelWidth);
//
//	// copy data back to host
//	BYTE * imageData_h = (BYTE*)malloc(imageSize);
//	hipMemcpy(imageData_h, imageData_d, imageSize, hipMemcpyDeviceToHost);
//
//	// construct the bitmap info header (DIB header)
//	BITMAPINFOHEADER bmpInfoHeader = { 0 };
//	bmpInfoHeader.biSize = sizeof(BITMAPINFOHEADER); // should be 40 bytes
//	bmpInfoHeader.biHeight = HEIGHT;
//	bmpInfoHeader.biWidth = WIDTH;
//	bmpInfoHeader.biPlanes = 1; // number of color planes (always 1)
//	bmpInfoHeader.biBitCount = pixelWidth * 8;
//	bmpInfoHeader.biCompression = BI_RGB; // do not compress
//	bmpInfoHeader.biSizeImage = imageSize; // image size in bytes
//	bmpInfoHeader.biClrUsed = 0; // no colors
//	bmpInfoHeader.biClrImportant = 0; // all colors important
//
//	// construct bitmap file header
//	BITMAPFILEHEADER bfh;
//	bfh.bfType = 0x4D42; // the first two bytes of the file are 'BM' in ASCII, in little endian
//	bfh.bfOffBits = sizeof(BITMAPINFOHEADER) + sizeof(BITMAPFILEHEADER) + (sizeof(RGBQUAD) * 256); // the offset (starting address of pixel data). size of headers + color table
//	bfh.bfSize = bfh.bfOffBits + bmpInfoHeader.biSizeImage; // total size of image including size of headers
//
//	// create the color table
//	RGBQUAD colorTable[256];
//	for (int i = 0; i < 256; i++)
//	{
//		colorTable[i].rgbBlue = (BYTE)i;
//		colorTable[i].rgbGreen = (BYTE)i;
//		colorTable[i].rgbRed = (BYTE)i;
//		colorTable[i].rgbReserved = (BYTE)i;
//	}
//
//	// write everything to file
//	ofstream imageFile;
//
//	char filePath[FILENAME_MAX];
//	// get the current working directory
//	if (!_getcwd(filePath, FILENAME_MAX))
//	{
//		printf("error accessing current working directory\n");
//		return 0;
//	}
//
//	printf("The current working directory is %s\n", filePath);
//	strcat_s(filePath, "\\mandelbrot.bmp"); // append the image file name
//
//	imageFile.open(filePath);
//	imageFile.write((char *)&bfh, sizeof(bfh)); // Write the File header
//	imageFile.write((char *)&bmpInfoHeader, sizeof(bmpInfoHeader)); // Write the bitmap info header
//	imageFile.write((char *)&colorTable, sizeof(RGBQUAD) * 256); // Write the color table
//
//	// if number of rows is a multiple of 4 bytes
//	if (WIDTH % 4 == 0)
//	{
//		// write the image judata
//		imageFile.write((char*)imageData_h, bmpInfoHeader.biSizeImage);
//	}
//	else
//	{
//		// else write and pad each row out with empty bytes
//		char* padding = new char[4 - WIDTH % 4];
//		for (int i = 0; i < HEIGHT; ++i)
//		{
//			imageFile.write((char *)&imageData_h[i * WIDTH], WIDTH);
//			imageFile.write((char *)padding, 4 - WIDTH % 4);
//		}
//	}
//
//	imageFile.close();
//	printf("image file saved to %s\n", filePath);
//
//	// clean up
//	hipDeviceReset();
//	hipFree(imageData_d);
//	free(imageData_h);
//
//	Mat imgjay = imread("mandelbrot.bmp");
//	namedWindow("MandelBrot View", 0);
//	imshow("MandelBrot View", imgjay);
//
//	waitKey(0);
//
//	return 0;
//}


#include "fractal.hpp"
#include <iostream>
#include <cmath>
#include <cstring>
#include <opencv2/opencv.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>

using namespace std;
using namespace cv;

#ifdef __HIPCC__

__global__ static void calculateMandelbrot(char *imageBuffer, double cx0, double cy0, double cx1, double cy1,
	int width, int height, int maxIter);

#define cudaCheck(ins) { _cudaCheck(ins, __FILE__, __LINE__); }

inline void _cudaCheck(hipError_t code, const char *file, int line)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "cudaCheck: %s %s %d\n", hipGetErrorString(code), file, line);
		exit(code);
	}
}

#else

static void calculateMandelbrotCPU(char *imageBuffer, double cx0, double cy0, double cx1, double cy1,
	int width, int height, int maxIter);

#endif

Fractal::Fractal(double cx0, double cy0, double cx1, double cy1, int width, int height, int maxIter)
{
	SetDimensions(cx0, cy1, cx1, cy1, maxIter);
	this->width = width;
	this->height = height;
	this->imageBuffer = new char[width * height * 3];
}

Fractal::~Fractal()
{
	delete[] this->imageBuffer;
}

void Fractal::SetDimensions(double cx0, double cy0, double cx1, double cy1, int maxIter)
{
	this->cx0 = cx0;
	this->cy0 = cy0;
	this->cx1 = cx1;
	this->cy1 = cy1;
	this->maxIter = maxIter;
}

char *Fractal::GetImageBuffer()
{

#ifdef __HIPCC__

	dim3 threadsPerBlock(16, 16);
	dim3 blocksPerGrid(this->width / threadsPerBlock.x, this->height / threadsPerBlock.y);

	char *imageBuffer_d;

	cudaCheck(hipMalloc(&imageBuffer_d, width * height * 3));
	calculateMandelbrot << <blocksPerGrid, threadsPerBlock >> > (imageBuffer_d, this->cx0, this->cy0, this->cx1, this->cy1,
		this->width, this->height, this->maxIter);

	cudaCheck(hipMemcpy(imageBuffer, imageBuffer_d, width * height * 3, hipMemcpyDeviceToHost));
	cudaCheck(hipFree(imageBuffer_d));

#else

	calculateMandelbrotCPU(this->imageBuffer, this->cx0, this->cy0, this->cx1, this->cy1,
		this->width, this->height, this->maxIter);

#endif

	return this->imageBuffer;
}

#ifdef __HIPCC__

__global__ static void calculateMandelbrot(char *imageBuffer, double cx0, double cy0, double cx1, double cy1,
	int width, int height, int maxIter)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int pixelId = (row * width + col) * 3;

	double x = 0, y = 0;
	double cx = (double)col / width * (cx1 - cx0) + cx0;
	double cy = (double)row / height * (cy0 - cy1) + cy1;

	int numberOfIterations = 0;
	double tempx;

	while ((x * x + y * y < 4.0) && (numberOfIterations <= maxIter))
	{
		tempx = x * x - y * y + cx;
		y = 2.0 * x * y + cy;
		x = tempx;
		numberOfIterations++;
	}

	int color = numberOfIterations;

	if (numberOfIterations == maxIter) color = 0;

	imageBuffer[pixelId] = 255 - color % 256;//color % 256;
	imageBuffer[pixelId + 1] = 0;
	imageBuffer[pixelId + 2] = color * 5 % 256;
}

#else

static void calculateMandelbrotCPU(char *imageBuffer, double cx0, double cy0, double cx1, double cy1,
	int width, int height, int maxIter)
{
	int nt, tid;
#pragma omp parallel private(tid)
	{
		nt = omp_get_num_threads();
		tid = omp_get_thread_num();
		int kt = (tid + 1) * (height / nt);
		int k = tid * (height / nt);
		//printf("nt: %d", nt);

		for (; k < kt; k++)
			for (int j = 0; j < width; j++)
			{
				int row = k;
				int col = j;
				int pixelId = (row * width + col) * 3;

				double x = 0, y = 0;
				double cx = (double)col / width * (cx1 - cx0) + cx0;
				double cy = (double)row / height * (cy0 - cy1) + cy1;

				int numberOfIterations = 0;
				double tempx;

				while ((x * x + y * y < 4.0) && (numberOfIterations <= maxIter))
				{
					tempx = x * x - y * y + cx;
					y = 2.0 * x * y + cy;
					x = tempx;
					numberOfIterations++;
				}

				int color = numberOfIterations;

				if (numberOfIterations == maxIter) color = 0;

				imageBuffer[pixelId] = 255 - color % 256;//color % 256;
				imageBuffer[pixelId + 1] = 0;
				imageBuffer[pixelId + 2] = color * 5 % 256;
			}
		//printf("thread %d finished\n", tid);
	}
}

#endif

int width = 1024, height = 1024, maxIter = 250;
double cx0 = -2, cy0 = -1.5, cx1 = 1, cy1 = 1.5, rangex, rangey;

Mat output(width, height, CV_8UC3);

bool clicked = false;
bool isImgCorrect = false;

double tempcx0, tempcy0, tempcx1, tempcy1;

void reset()
{
	cx0 = -2; cy0 = -1.5; cx1 = 1; cy1 = 1.5;
	maxIter = 250;
	isImgCorrect = false;
}

void mouseCallBack(int event, int x, int y, int flags, void *userdata);

int main(int argc, char *argv[])
{
	Fractal mandelbrot(cx0, cy0, cx1, cy1, width, height, maxIter);

	namedWindow("MandelWindow", 1);

	setMouseCallback("MandelWindow", mouseCallBack);

	for (;;)
	{
		if (!isImgCorrect)
		{
			mandelbrot.SetDimensions(cx0, cy0, cx1, cy1, maxIter);
			memcpy(output.data, mandelbrot.GetImageBuffer(), width * height * 3);

			imshow("MandelWindow", output);

			isImgCorrect = true;
		}
		int key = waitKey(5);

		switch (key)
		{
		case 'q':
			return 0;
		case 'k':
			rangey = cy1 - cy0;
			cy0 += ((double)1 / 5) * rangey;
			cy1 += ((double)1 / 5) * rangey;
			isImgCorrect = false;
			break;
		case 'j':
			rangey = cy1 - cy0;
			cy0 -= ((double)1 / 5) * rangey;
			cy1 -= ((double)1 / 5) * rangey;
			isImgCorrect = false;
			break;
		case 'l':
			rangex = cx1 - cx0;
			cx0 += ((double)1 / 5) * rangex;
			cx1 += ((double)1 / 5) * rangex;
			isImgCorrect = false;
			break;
		case 'h':
			rangex = cx1 - cx0;
			cx0 -= ((double)1 / 5) * rangex;
			cx1 -= ((double)1 / 5) * rangex;
			isImgCorrect = false;
			break;
		case 'z':
			rangex = cx1 - cx0;
			rangey = cy1 - cy0;
			tempcx0 = cx1 - ((double)4 / 5) * rangex;
			tempcy0 = cy1 - ((double)4 / 5) * rangey;
			cx1 = cx0 + ((double)4 / 5) * rangex;
			cy1 = cy0 + ((double)4 / 5) * rangey;
			cx0 = tempcx0;
			cy0 = tempcy0;
			//cout << "z pressed" << endl;
			isImgCorrect = false;
			break;
		case 'u':
			rangex = cx1 - cx0;
			rangey = cy1 - cy0;
			tempcx0 = cx1 - ((double)5 / 4) * rangex;
			tempcy0 = cy1 - ((double)5 / 4) * rangey;
			cx1 = cx0 + ((double)5 / 4) * rangex;
			cy1 = cy0 + ((double)5 / 4) * rangey;
			cx0 = tempcx0;
			cy0 = tempcy0;
			//cout << "z pressed" << endl;
			isImgCorrect = false;
			break;
		case 'i':
			maxIter *= 2;
			cout << "number of iterations: " << maxIter << endl;
			isImgCorrect = false;
			break;
		case 'd':
			maxIter /= 2;
			cout << "number of iterations: " << maxIter << endl;
			isImgCorrect = false;
			break;
		case 'r':
			reset();
			break;
		}
	}
	return 0;
}

int rx0, ry0, rx1, ry1;

void mouseCallBack(int event, int x, int y, int flags, void *userdata)
{
	if (event == EVENT_LBUTTONDOWN)
	{
		//cout << "LButtonDown x: " << x << "\ty: " << y << endl;
		rx0 = x; ry0 = y;
		clicked = true;
	}

	if (event == EVENT_LBUTTONUP)
	{
		//cout << "LButtonUp x: " << x << "\ty: " << y << endl;
		rangex = cx1 - cx0;
		rangey = cy1 - cy0;
		tempcx0 = (double)rx0 / width * rangex + cx0;
		tempcy0 = (double)(height - ry0) / height * rangey + cy0;
		tempcx1 = (double)rx1 / width * rangex + cx0;
		tempcy1 = (double)(height - ry1) / height * rangey + cy0;
		cx0 = min(tempcx0, tempcx1); cx1 = max(tempcx0, tempcx1);
		cy0 = min(tempcy0, tempcy1); cy1 = max(tempcy0, tempcy1);
		//cout << cx0 << ' ' << cy0 << ' ' << cx1 << ' ' << cy1 << endl;
		isImgCorrect = false;
		clicked = false;
	}

	if (event == EVENT_MOUSEMOVE && clicked)
	{
		//cout << "MouseMove x: " << x << "\ty: " << y << endl;
		Mat outputTemp = output.clone();

		rx1 = x; ry1 = ry0 + (((ry0 - y) < 0) ? 1 : -1) * abs(rx0 - x);
		rectangle(outputTemp, Point(rx0, ry0), Point(rx1, ry1), Scalar(255, 255, 255));

		imshow("MandelWindow", outputTemp);
	}
}
